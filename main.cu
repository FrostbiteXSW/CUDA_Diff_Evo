#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <chrono>
#include <hiprand.h>
#include "utils.h"

using namespace std;
using namespace chrono;

#define BLOCK_NUM  64		// GPU������
#define BLOCK_SIZE 512		// GPU���С
#define RAND_SIZE  1000		// CUDA��������д�С
#define ITERATIONS 1000		// ��ֽ�������
#define ARG_NUM	   10		// ��������
#define ARG_LIMIT  100		// ��������ֵ��Χ����
#define BIAS	   22		// ƫ��
#define F		   0.5		// ��������
#define CR		   0.3		// �������

/*
 * �ⷽ�����磺
 * 
 *		��(-1)^i * (x_i)^i * i  + BIAS = 0, i > 0
 *		
 *	���У�x_i�ľ���ֵС�ڵ���ARG_LIMIT
 *	ARG_NUM������������������i��
 */


/**
 * \brief ʹ��GPU���в�ֽ����������Ӵ��Ĳ�����
 * \param arg_list ��ǰ���Ų����б�
 * \param result_list GPU����õ��������Ӵ�����������
 * \param rand Ԥ����������б�
 */
__global__ void DifferentialEvolution(const double* arg_list, double* result_list, const double* rand) {
	// GPU������������
	__shared__ double results[BLOCK_SIZE][ARG_NUM + 1];

	// ������±꼰����
	auto randIndex = threadIdx.x + 1;
	const auto step = blockIdx.x + 1;
	
	// ����
	for (auto i = 0; i < ARG_NUM; ++i) {
		int r1, r2, r3;
		do {
			r1 = int(rand[randIndex] * ARG_NUM) % ARG_NUM;
			randIndex = (randIndex + step) % RAND_SIZE;
			r2 = int(rand[randIndex] * ARG_NUM) % ARG_NUM;
			randIndex = (randIndex + step) % RAND_SIZE;
			r3 = int(rand[randIndex] * ARG_NUM) % ARG_NUM;
			randIndex = (randIndex + step) % RAND_SIZE;
		}
		while (r1 == r2 || r2 == r3 || r1 == r3);
		results[threadIdx.x][i] = arg_list[r1] + F * (arg_list[r2] - arg_list[r3]);
		if (abs(results[threadIdx.x][i]) > ARG_LIMIT) {
			results[threadIdx.x][i] = (rand[randIndex] - 0.5) * 2 * ARG_LIMIT;
			randIndex = (randIndex + step) % RAND_SIZE;
		}
	}
	
	// ����
	const auto j = int(rand[randIndex] * ARG_NUM) % ARG_NUM;
	randIndex = (randIndex + step) % RAND_SIZE;
	for (auto i = 0; i < ARG_NUM; ++i) {
		if (i != j && rand[randIndex] > CR) {
			results[threadIdx.x][i] = arg_list[i];
		}
		randIndex = (randIndex + step) % RAND_SIZE;		
	}

	// ����
	results[threadIdx.x][ARG_NUM] = 0;
	for (auto i = 0; i < ARG_NUM; ++i) {
		auto temp = (i + 1.) * ((i + 1) % 2 == 0 ? 1 : -1);
		for (auto n = 0; n < i + 1; ++n) {
			temp *= results[threadIdx.x][i];
		}
		results[threadIdx.x][ARG_NUM] += temp;
	}
	results[threadIdx.x][ARG_NUM] += BIAS;
	__syncthreads();

	// ѡ��
	if (threadIdx.x == 0) {
		for (auto i = 1; i < BLOCK_SIZE; ++i) {
			if (abs(results[i][ARG_NUM]) < abs(results[0][ARG_NUM])) {
				for (auto n = 0; n < ARG_NUM + 1; ++n) {
					results[0][n] = results[i][n];
				}
			}
		}
		for (auto i = 0; i < ARG_NUM + 1; ++i) {
			result_list[blockIdx.x * (ARG_NUM + 1) + i] = results[0][i];
		}
	}
}


/**
 * \brief ��GPU�Ͻ����Ӵ�ѡ�񣬽�ʹ�õ��߳�
 * \param arg_list ��ǰ���Ų����б�
 * \param result_list GPU����õ��������Ӵ�����������
 */
__global__ void SelectNextGeneration(double* arg_list, const double* result_list) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {		
		auto bestResult = -1;
		for (auto j = 0; j < BLOCK_NUM; ++j) {
			if (abs(result_list[j * (ARG_NUM + 1) + ARG_NUM]) < abs(arg_list[ARG_NUM])) {
				bestResult = j;
			}
		}			
		if (bestResult >= 0) {
			memcpy(arg_list, &result_list[bestResult * (ARG_NUM + 1)], sizeof(double) * (ARG_NUM + 1));
		}
	}
}

int main() {
	// ��ǰ���Ų����б�������[argv], result��
	const auto hostArgList = static_cast<double*>(malloc(sizeof(double) * (ARG_NUM + 1)));

	// ���Ų����б���GPU�洢�еĻ�����
	double* deviceArgList;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&deviceArgList), sizeof(double) * (ARG_NUM + 1)));
	
	// GPU����õ��������Ӵ�����������
	double* deviceResultList;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&deviceResultList), sizeof(double) * BLOCK_NUM * (ARG_NUM + 1)));

	// ��ʼ����Ⱥ
	srand(time(nullptr));
	for (auto i = 0; i < ARG_NUM; ++i) {
		hostArgList[i] = (double(rand()) / RAND_MAX - 0.5) * 2 * ARG_LIMIT;
	}
	hostArgList[ARG_NUM] = 0.;
	for (auto i = 0; i < ARG_NUM; ++i) {
		auto temp = (i + 1.) * ((i + 1) % 2 == 0 ? 1 : -1);
		for (auto n = 0; n < i + 1; ++n) {
			temp *= hostArgList[i];
		}
		hostArgList[ARG_NUM] += temp;
	}
	hostArgList[ARG_NUM] += BIAS;

	// ��ʼ��CUDA�������������������
	double *deviceRand1, *deviceRand2;
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&deviceRand1), sizeof(double) * RAND_SIZE));
	checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&deviceRand2), sizeof(double) * RAND_SIZE));
    hiprandGenerator_t deviceRandGenerator;
	hiprandCreateGenerator(&deviceRandGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(deviceRandGenerator, time(nullptr));
	hiprandGenerateUniformDouble(deviceRandGenerator, deviceRand1, RAND_SIZE);
	
	// ��ֽ���	
	checkCudaErrors(hipMemcpy(deviceArgList, hostArgList, sizeof(double) * (ARG_NUM + 1), hipMemcpyHostToDevice));
	const auto start = system_clock::now();	
	for (auto i = 0; i < ITERATIONS; ++i) {
		// GPU���������Ӵ����
		DifferentialEvolution<<<BLOCK_NUM, BLOCK_SIZE>>>(deviceArgList, deviceResultList, i % 2 ? deviceRand2 : deviceRand1);
		
		// �����������������
		hiprandGenerateUniformDouble(deviceRandGenerator, i % 2 ? deviceRand1 : deviceRand2, RAND_SIZE);
		
		// �����Ӵ�ѡ��
		hipDeviceSynchronize();
		SelectNextGeneration<<<1, 1>>>(deviceArgList, deviceResultList);
	}
	const auto elapsedTime = duration_cast<milliseconds>(system_clock::now() - start).count();
	printf("Algorithm running time is %lld ms\n", elapsedTime);
	checkCudaErrors(hipMemcpy(hostArgList, deviceArgList, sizeof(double) * (ARG_NUM + 1), hipMemcpyDeviceToHost));

	// ������
	for (auto i = 0; i < ARG_NUM; ++i) {
		printf("x%d = %f\n", i + 1, hostArgList[i]);
	}
	printf("Result = %f\n", hostArgList[ARG_NUM]);

	// ���Խ��
	auto realResult = 0.;
	for (auto i = 0; i < ARG_NUM; ++i) {
		realResult += pow(-1, i + 1) * pow(hostArgList[i], i + 1) * (i + 1);
	}
	printf("Validating Result = %f\n", realResult + BIAS);
	
	// �ͷ�CPU�洢
	free(hostArgList);

	// �ͷ�GPU�洢
	checkCudaErrors(hipFree(deviceRand1));
	checkCudaErrors(hipFree(deviceRand2));
	checkCudaErrors(hipFree(deviceArgList));
	checkCudaErrors(hipFree(deviceResultList));
}
